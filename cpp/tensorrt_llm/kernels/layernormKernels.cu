#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/layernormKernels.h"

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

// Compute layer normalization for a single value
template <typename Tf, typename T>
__inline__ __device__ Tf compute_layernorm(Tf val, float s_mean, float s_variance, const T* gamma, const T* beta, int i)
{
    Tf ret = (val - s_mean) * s_variance * cuda_cast<Tf>(gamma[i]);
    if (beta != nullptr)
    {
        ret = ret + cuda_cast<Tf>(beta[i]);
    }
    return ret;
}

// General layer normalization kernel for a given data type and optionally using a difference-of-squares method
template <typename T, bool USE_DIFF_OF_SQUARES = false>
__global__ void generalLayerNorm(const T* input, const T* gamma, const T* beta, T* normed_output, const float eps,
    int tokens, int hidden_dim, const float* scale_orig_quant_per_tensor, float* scale_orig_quant_per_token,
    int8_t* normed_output_quant, bool use_shmem)
{
    // Shared memory and variables for mean and variance computation
    extern __shared__ __align__(sizeof(float)) char _shmem[];
    T* shmem = reinterpret_cast<T*>(_shmem);
    __shared__ float s_mean;
    __shared__ float s_variance;

    // Thread and block indices
    const int tidx = threadIdx.x;
    const int bidx = blockIdx.x;

    // Variables for mean and variance computation
    float mean = 0.0f;
    float variance = 0.0f;
    float local_sum = 0.0f;
    float local_var_sum = 0.0f;

    // Loop over the elements in the row
    const int n_elems = hidden_dim / num_elems<T>::value;
    for (int i = tidx; i < n_elems; i += blockDim.x)
    {
        // Load the value from input or shared memory
        const T val = use_shmem ? shmem[i] : input[bidx * n_elems + i];

        // Convert the value to float and accumulate the sum and sum of squares
        const float_packed_t val_f = cuda_cast<float_packed_t>(val);
        local_sum += cuda_sum<float>(val_f);
        if (USE_DIFF_OF_SQUARES)
        {
            local_var_sum += cuda_sum<float>(val_f * val_f);
        }
    }

    // Reduce the sum and sum of squares across the block
    if (USE_DIFF_OF_SQUARES)
    {
        float packed[2] = {local_sum, local_var_sum};
        blockReduceSumV2<float, 2>(packed);
        mean = packed[0];
        variance = packed[1];
    }
    else
    {
        mean = blockReduceSum(local_sum);
    }

    // Compute the mean and variance if not using the difference-of-squares method
    if (threadIdx.x == 0)
    {
        mean = mean / hidden_dim;
        s_mean = mean;
        if (USE_DIFF_OF_SQUARES)
        {
            variance = (variance / hidden_dim) - (mean * mean); // Var[x] = E[x²] - E[x]²
            s_variance = rsqrtf(variance + eps);
        }
    }
    __syncthreads();

    // Compute the variance if using the difference-of-squares method
    if (!USE_DIFF_OF_SQUARES)
    {
        for (int i = tidx; i < n_elems; i += blockDim.x)
        {
            // Load the value from input or shared memory
            const T val = use_shmem ? shmem[i] : input[bidx * n_elems + i];

            // Compute the difference from the mean and accumulate the sum of squares
            float_packed_t diff = cuda_cast<float_packed_t>(val) - s_mean;
            local_var_sum += cuda_sum<float>(diff * diff);
        }
        variance = blockReduceSum(local_var_sum);

        // Compute the variance and scale
        if (threadIdx.x == 0)
        {
            s_variance = rsqrtf(variance / hidden_dim + eps);
        }
        __syncthreads();
    }

    // Compute the output value and store it in the output array
    const bool with_per_token_scaling = scale_orig_quant_per_token != nullptr;
    const bool with_per_tensor_scaling = scale_orig_quant_per_tensor != nullptr;
    const float_packed_t
