#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/decoderMaskedMultiheadAttentionUtils.h"
#include "tensorrt_llm/kernels/gptKernels.h"
#include "tensorrt_llm/kernels/kvCacheUtils.h"
#include "tensorrt_llm/kernels/unfusedAttentionKernels.h"

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

__inline__ __device__ int target_index(int id1, int id2, int id3, int id4, int dim_1, int dim_2, int dim_3, int dim_4)
{
    return id1 * (dim_2 * dim_3 * dim_4) + id3 * (dim_2 * dim_4) + id2 * dim_4 + id4;
}

template <typename T>
__global__ void addQKVBiasIA3Transpose(T* q_out, T* k_out, T* v_out, const T* __restrict q_in,
    const T* __restrict bias_q, const T* __restrict k_in, const T* __restrict bias_k, const T* __restrict v_in,
    const T* __restrict bias_v, const int* ia3_tasks, const T* ia3_key_weights, const T* ia3_value_weights,
    const int batch_size, const int seq_len, const int head_num, const int size_per_head)
{
    const int n = head_num * size_per_head;
    const int batch_id = blockIdx.x;
    const int word_id = blockIdx.y;
    const int row_id = batch_id * seq_len + word_id;

    const bool use_ia3 = ia3_tasks != nullptr;
    const int ia3_task = use_ia3 ? ia3_tasks[batch_id] : 0;
    const bool use_ia3_key = use_ia3 && (ia3_key_weights != nullptr);
    const bool use_ia3_value = use_ia3 && (ia3_value_weights != nullptr);

    for (int col_id = threadIdx.x; col_id < n; col_id += blockDim.x)
    {
        const int head_id = col_id / size_per_head;
        const int size_id = col_id % size_per_head;
        const int target_id = batch_id * (head_num * seq_len * size_per_head) + head_id * seq_len * size_per_head
            + word_id * size_per_head + size_id;
        const int src_id = row_id * n + col_id;

        T q = ldg(&q_in[src_id]);
        q_out[target_id] = add(q, ldg(&bias_q[col_id]));

        T k = add(ldg(&k_in[src_id]), ldg(&bias_k[col_id]));
        if (use_ia3_key)
        {
            k = k * ia3_key_weights[ia3_task * n + col_id];
        }
        k_out[target_id] = k;

        T v = add(ldg(&v_in[src_id]), ldg(&bias_v[col_id]));
        if (use_ia3_value)
        {
            v = v * ia3_value_weights[ia3_task * n + col_id];
        }
        v_out[target_id] = v;
    }
}

template <typename T>
__global__ void QKVIA3Transpose(T* q_out, T* k_out, T* v_out, const T* __restrict q_in, const T* __restrict k_in,
    const T* __restrict v_in, const int* ia3_tasks, const T* __restrict ia3_key_weights,
    const T* __restrict ia3_value_weights, const int batch_size, const int seq_len, const int head_num,
    const int size_per_head)
{
    const int n = head_num * size_per_head;
    const int batch_id = blockIdx.x;
    const int word_id = blockIdx.y;
    const int row_id = batch_id * seq_len + word_id;

    const bool use_ia3 = ia3_tasks != nullptr;
    const int ia3_task = use_ia3 ? ia3_tasks[batch_id] : 0;
    const bool use_ia3_key = use_ia3 && (ia3_key_weights != nullptr);
    const bool use_ia3_value = use_ia3 && (ia3_value_weights != nullptr);

    for (int col_id = threadIdx.x; col
