#include "hip/hip_runtime.h"



#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/common/stringUtils.h"
#include "tensorrt_llm/kernels/beamSearchTopkKernels.h"

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

template <typename T>
__device__ __forceinline__ T apply_length_penalty(T log_prob, int length, float length_penalty)
{
    // score = log(prob) / (length)^length_penalty.
    if (length_penalty == 0.0f || length == 1)
    {
        return log_prob;
    }
    return log_prob / static_cast<T>(powf((float) length, length_penalty));
}

template <typename T, int MAX_K, int THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE) __global__
    void beam_topK_kernel(const T* log_probs, int* topk_tmp_id_buf, T* topk_tmp_val_buf, const bool* finished,
        const int* sequence_lengths, const int vocab_size, T diversity_rate, float length_penalty)
{
    typedef hipcub::BlockReduce<TopK<T, MAX_K>, THREADBLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    int thread_id = threadIdx.x;
    int block_id = blockIdx.x; // batch beam index.
    TopK<T, MAX_K> partial;

    const bool IS_FP16 = std::is_same<T, half>::value;
    const T MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

#pragma unroll
    for (int i = 0; i < MAX_K; ++i)
    {
        partial.p[i] = -1;
        partial.u[i] = -MAX_T_VAL;
    }

#pragma unroll
    for (int elem_id = thread_id; elem_id < vocab_size; elem_id += THREADBLOCK_SIZE)
    {
        int index = elem_id + block_id * vocab_size;
        T score = length_penalty == 0.0f
            ? log_probs[index]
            : apply_length_penalty(log_probs[index],
                finished[block_id] ? sequence_lengths[block_id] : sequence_lengths[block_id] + 1, length_penalty);
        partial.insert(score, index);
    }

    TopK<T, MAX_K> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_op<T, MAX_K>);

    if (thread_id == 0)
    {
        int index = block_id * MAX_K;

#pragma unroll
        for (int i = 0; i < MAX_K; ++i)
        {
            topk_tmp_id_buf[index + i] = total.p[i];
            topk_tmp_val_buf[index + i] = total.u[i] + diversity_rate * (T) i;
        }
    }
}

template <typename T, int MAX_K, int THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE) __global__
    void batch_topK_kernel(int* topk_tmp_id_buf, T* topk_tmp_val_buf, int* id_buf)
{
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    const bool IS_FP16 = std::is_same<T, half>::value;
    const T MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;
    TopK<T, MAX_K> partial;
    if (thread_id == 0)
    {
        for (int i = 0; i < MAX_K; ++i)
        {
            partial.p[i] = -1;
            partial.u[i] = -MAX_T_VAL;
        }

        int index = block_id * MAX_K * MAX_K;
        for (int i = 0; i < MAX_K * MAX_K; i++)
        {
            partial.insert((T) topk_tmp_val_buf[index + i], topk_tmp_id_buf[index + i]);
        }

        index = block_id * MAX_K;
        for (int i = 0; i < MAX_K; i++)
        {
            id_buf[index + i] = partial.p[i];
        }
    }
}

template <typename T, int MAX_K, int THREADBLOCK_SIZE
