#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaBf16Wrapper.h"
#include "tensorrt_llm/common/cudaFp8Utils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/gptKernels.h"
#include <hipcub/hipcub.hpp>

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

// A stateful callback functor that maintains the running sum between consecutive scans.
struct BlockPrefixCallbackOp
{
    // Running prefix
    int mRunningTotal;

    // Constructor
    __device__ BlockPrefixCallbackOp(int runningTotal)
        : mRunningTotal(runningTotal)
    {
    }

    // Thread-0 is responsible for returning a value for seeding the block-wide scan.
    __device__ int operator()(int blockAggregate)
    {
        int oldPrefix = mRunningTotal;
        mRunningTotal += blockAggregate;
        return oldPrefix;
    }
};

// Given an array of sequence lengths, with batchSize elements, that kernel computes the exclusive
// prefix-sums of the sequence lengths. There are (batchSize+1) elements in seqOffsets.
//
// seqOffsets[ 0]        = 0
// seqOffsets[ii]        = seqLengths[0] + .. + seqLengths[ii-1],
// seqOffsets[batchSize] = seqLengths[0] + .. + seqLengths[batchSize-1]
//
// This kernel uses a single thread block of THREADS_PER_BLOCK threads.

template <int THREADS_PER_BLOCK>
__global__ __launch_bounds__(THREADS_PER_BLOCK) void computeSeqOffsets(
    int* seqOffsets, const int* seqLengths, int batchSize)
{
    // The implementation of the parallel scan in the thread block (see CUB for details).
    using BlockScan = hipcub::BlockScan<int, THREADS_PER_BLOCK>;

    // Allocate storage in shared memory to do the scan.
    __shared__ typename BlockScan::TempStorage tempStorage;

    // This prefixOp operator keeps a running sum for when we need multiple iterations of the loop.
    BlockPrefixCallbackOp prefixOp(0);

    // Iterate over the sequences in the batch.
    //
    // The loop index does not depend on the thread index to make sure all the threads enter the
    // loop as we have __syncthreads in it (and we need all threads to participate to avoid
    // deadlocks).
    for (int batchOffset = 0; batchOffset <= batchSize; batchOffset += THREADS_PER_BLOCK)
    {
        // The index of the batch.
        int batchIdx = batchOffset + threadIdx.x;

        // Threads that correspond to valid sequences read the length.
        int seqLength = 0;
        if (batchIdx < batchSize)
        {
            seqLength = seqLengths[batchIdx];
        }

        // Do the prefix-scan (it calls syncthreads internally).
        int seqOffset;
        BlockScan(tempStorage).ExclusiveSum(seqLength, seqOffset, prefixOp);

        // Store the result.
        if (batchIdx <= batchSize)
        {
            seqOffsets[batchIdx] = seqOffset;
        }

        // Make sure the shared memory can be reused for the next iteration of the loop.
        __syncthreads();
    }
}

// This kernel computes the padding offsets: Given the index (idx) of a token in a ragged tensor,
// we need the index of the token in the corresponding tensor with padding. We compute an array
// of numTokens elements, called the paddingOffsets, such that the position in the padded tensor
// of the token "idx" in the ragged tensor is given by idx + paddingOffset[idx].
//
// That kernel uses a grid of batchSize blocks.

__global__ void computePaddingOffsets(int* paddingOffsets, const int* seqOffsets, int maxSeqLength)
{
    // The index of the sequence in the batch.
    int batchIdx = blockIdx.x;

    // The beginning of the sequence.
    int seqBegin = seqOffsets[batchIdx];
    // The offset to the 1st element of the next sequence.
    int seqEnd = seqOffsets[batchIdx + 1];
    // The length of the sequence.
    int seqLength = seqEnd - seqBegin;

    // The number of padded tokens in the previous sequences.
    int paddingOffset = batchIdx * maxSeqLength - seqBegin;

    // Iterate over the tokens to update the number of padded elements.
    for (int tokenIdx = threadIdx.x; tokenIdx < seqLength; tokenIdx += blockDim.x)
    {
        paddingOffsets[seqBegin + tokenIdx] = paddingOffset + max(0, tokenIdx - seqLength);
    }
}

// This kernel computes the attention mask. We must compute this on-the-fly in the future.

template <typename AttentionMaskDataType>
__global__ void computeAttentionMask(
    AttentionMaskDataType* attentionMask, const int* seqOffsets, int maxSeqLength, AttentionMaskType attentionMaskType)
{
    // The index of the sequence in the batch.
    int batchIdx = blockIdx.y;

    // The number of items in
